/*Copyright (C) 2022 - Pénélope Delabrière, <penelope.delabriere@master-developpement-logiciel.fr>
This program is free software: you can redistribute it and/or modify it under the terms of the GNU General Public License as published by the Free Software Foundation, either version 3 of the License, or (at your option) any later version.

This program is distributed in the hope that it will be useful, but WITHOUT ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU General Public License for more details.*/

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void kreduce(float *d_vec, float *d_sum, int size){
    int index = blockIdx.x*blockDim.x+threadIdx.x;
    for (int i = size / 2; i > 0; i /= 2){
        if( index < i ){
            d_vec[index] += d_vec[index+i];
        }

        __syncthreads();
    }
    if(index == 0){
        d_sum[blockIdx.x] = d_vec[blockIdx.x+threadIdx.x];
    }
}


void reduce(float *vec, float *sum, int size){
    float *d_vec;
    float *d_sum;
    float *d_sum2;
    int bytes = size*sizeof(float);

    hipMalloc((void **)&d_vec, bytes);
    hipMalloc((void **)&d_sum, bytes);
    hipMalloc((void **)&d_sum2, sizeof(float));
    hipMemcpy(d_vec, vec, bytes, hipMemcpyHostToDevice);

    kreduce<<<(size+1024-1)/1024, 1024>>>(d_vec, d_sum, size);
    kreduce<<<1, 1024>>>(d_sum, d_sum2, 1024);

    hipMemcpy(sum, d_sum2, sizeof(float), hipMemcpyDeviceToHost);
    hipFree(d_vec);
    hipFree(d_sum);
    hipFree(d_sum2);
}

int main(int argc, char **argv){
    int size = 4096;
    float *vec = (float*) malloc(size*sizeof(float));
    float *sum = (float*) malloc(size*sizeof(float));

    for(int i = 0; i < size; i++){
        vec[i] = 1;
    }

    for(int i = 0; i < size; i++){
        sum[i] = 0;
    }

    reduce(vec, sum, size);

    printf("%f", sum[0]);


}
