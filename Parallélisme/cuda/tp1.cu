// -----------------------------------------------------------
// (C) 2022 Pénélope Delabrière, Toulouse, France
// Released under GNU Affero General Public License v3.0 (AGPLv3)
// -----------------------------------------------------------

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void kreduce(float *d_vec, float* d_sum, int size){
  int index = blockIdx.x * blockDim.x+threadIdx.x;
  for (int i = size / 2; i > 0; i /= 2){
    if( index < i ){
      d_vec[index] += d_vec[index+i];
    }
    __syncthreads();
  }
  if(index%blockDim.x==0){
    d_sum[blockIdx.x] = d_vec[blockIdx.x*blockDim.x];
  }
}


void reduce(float *vec, float *sum, int size){
  float *d_vec;
  float *d_sum;
  float *d_sum2;
  int bytes = size*sizeof(float);

  hipMalloc((void **)&d_vec, bytes);
  hipMalloc((void **)&d_sum, sizeof(float));
  hipMalloc((void **)&d_sum2, sizeof(float));
  hipMemcpy(d_vec, vec, bytes, hipMemcpyHostToDevice);

  kreduce<<<(size+1024-1)/1024, 1024>>>(d_vec, d_sum, size);
  kreduce<<<1, 1024>>>(d_sum, d_sum2, 1024);

  hipMemcpy(sum, d_sum2, sizeof(float), hipMemcpyDeviceToHost);
  hipFree(d_vec);
  hipFree(d_sum);
  hipFree(d_sum2);
}

int main(int argc, char **argv){
  int size = 1024;
  float *vec = (float*) malloc(size*sizeof(float));
  float *sum = (float*) malloc(size*sizeof(float));

  for(int i = 0; i < size; i++){
    vec[i] = 1;
  }

  for(int i = 0; i < size; i++){
    sum[i] = 0;
  }

  reduce(vec, sum, size);

  for (int i = 0; i < size; i++){
    printf("%f", sum[i]);
  }

}
