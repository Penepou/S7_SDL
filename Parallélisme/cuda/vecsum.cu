/*Copyright (C) 2022 - Pénélope Delabrière, <penelope.delabriere@master-developpement-logiciel.fr>
This program is free software: you can redistribute it and/or modify it under the terms of the GNU General Public License as published by the Free Software Foundation, either version 3 of the License, or (at your option) any later version.

This program is distributed in the hope that it will be useful, but WITHOUT ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU General Public License for more details.*/

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <string.h>
#include <assert.h>

__global__ void kreduce(float *vec, int size)
{
	int index = threadIdx.x;
	for (int i = size / 2; i > 0; i /= 2)
	{
		if (index < i)
		{
			vec[index] += vec[i + index];
		}
		__syncthreads();
	}
	return;
}

void reduce(float *vec, float *sum, int size)
{
	float *d_vec;
	int bytes = size * sizeof(float);
	hipMalloc((void **)&d_vec, bytes);
	hipMemcpy(d_vec, vec, bytes, hipMemcpyHostToDevice);
	kreduce<<<1, size>>>(d_vec, size);
	hipMemcpy(sum, d_vec, sizeof(float), hipMemcpyDeviceToHost);
	hipFree(d_vec);
	return;
}

int main(int argc, char **argv)
{
	float *vec, *sum;
	int size = 1024, incr;

	vec = (float *)malloc(size * sizeof(float));
	sum = (float *)malloc(size * sizeof(float));

	for (incr = 0; incr < size; incr++)
	{
		vec[incr] = 1.;
	}

	reduce(vec, sum, size);
	for (int i = 0; i < size; i++)
		printf("%f\n", sum[i]);

	free(vec);
	free(sum);
	return;
}

/*
int main(int argc, char **argv){
  if (argc < 2){
	printf("Usage: <filename>\n");
	exit(-1);
  }
  int size;
  float sum[1024];
  float *vec;
  FILE *f = fopen(argv[1],"r");
  fscanf(f,"%d\n",&size);
  size = 1 << size;
  if (size >= (1 << 20)){
	printf("Size (%u) is too large: size is limited to 2^20\n",size);
	exit(-1);
  }
   vec = (float *) malloc(size * sizeof(float)); assert(vec);
  for (int i=0; i<size; i++){
	fscanf(f, "%f\n",&(vec[i]));
  }

  for (int i=0; i<size; i++){
	sum[i]=0.;
  }
  reduce(vec, sum, size);
  printf("sum = %f\n", sum[0]);
  fclose(f);
}*/
